#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 16
__global__ void mandelKernel(int* d_img, float lowerX, float lowerY, float stepX, float stepY, int width, int height, int maxIterations) 
{
    // To avoid error caused by the floating number, use the following pseudo code
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    unsigned int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    if (thisX < width && thisY < height) {
        int idx = thisY * width + thisX;
        float c_re = lowerX + thisX * stepX;
        float c_im = lowerY + thisY * stepY;
        float z_re = c_re, z_im = c_im;
        int i = 0;
        for (i = 0; i < maxIterations; ++i)
        {
            if (z_re * z_re + z_im * z_im > 4.f)
                break;

            float new_re = z_re * z_re - z_im * z_im;
            float new_im = 2.f * z_re * z_im;
            z_re = c_re + new_re;
            z_im = c_im + new_im;
        }
        d_img[idx] = i;
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int* d_img, *host_img;
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    hipMalloc((void **)&d_img, resX * resY * sizeof(int));
    host_img = (int *) malloc(resX * resY * sizeof(int)); // kernel1
    // cudaHostAlloc((void **)&host_img, resX * resY * sizeof(int),cudaHostAllocDefault); // kernel2
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock(resX / BLOCK_SIZE, resY / BLOCK_SIZE);
    mandelKernel<<<numBlock, blockSize>>>(d_img, lowerX, lowerY, stepX, stepY, resX, resY, maxIterations);
    
    hipDeviceSynchronize();
    hipMemcpy(host_img, d_img, resX * resY * sizeof(int), hipMemcpyDeviceToHost);
    memcpy(img,host_img,resX * resY * sizeof(int));
    hipFree(d_img);
}
