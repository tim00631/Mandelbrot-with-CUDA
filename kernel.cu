#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define BLOCK_SIZE 16
__global__ void mandelKernel(int* d_img, float lowerX, float lowerY, float stepX, float stepY, int width, int height, int maxIterations, int g_width, int g_height) 
{
    // To avoid error caused by the floating number, use the following pseudo code
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

    unsigned int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    // size_t g_width = 16;
    // size_t g_height = 16;
    size_t end_j = thisY + g_height;
    for (int j = thisY; j < end_j; j++)
    {
        for (int i = thisX; i < g_width; ++i)
        {
            if (j < width && i < height) {
                int idx = j * width + i;
                float c_re = lowerX + i * stepX;
                float c_im = lowerY + j * stepY;
                float z_re = c_re, z_im = c_im;
                int val = 0;
                for (val = 0; i < maxIterations; ++val)
                {
                    if (z_re * z_re + z_im * z_im > 4.f)
                        break;
        
                    float new_re = z_re * z_re - z_im * z_im;
                    float new_im = 2.f * z_re * z_im;
                    z_re = c_re + new_re;
                    z_im = c_im + new_im;
                }
                d_img[idx] = val;
            }
        }
    }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    int* d_img, *host_img;
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;
    // cudaMalloc((void **)&d_img, resX * resY * sizeof(int)); // kernel1
    // host_img = (int *) malloc(resX * resY * sizeof(int)); // kernel1
    size_t pitch; // kernel2
    hipMallocPitch((void **)&d_img, &pitch, sizeof(float)*resX, resY); // kernel2
    hipHostAlloc((void **)&host_img, resX * resY * sizeof(int),hipHostMallocDefault); // kernel2
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlock(resX / BLOCK_SIZE, resY / BLOCK_SIZE);
    size_t g_width = resX / numBlock.x / BLOCK_SIZE;
    size_t g_height = resY /numBlock.y / BLOCK_SIZE;

    mandelKernel<<<numBlock, blockSize>>>(d_img, lowerX, lowerY, stepX, stepY, resX, resY, maxIterations, g_width, g_height);
    printf("%d, %d\n",numBlock.x, numBlock.y);
    printf("%d, %d\n",g_width, g_height);
    hipDeviceSynchronize();
    hipMemcpy(host_img, d_img, resX * resY * sizeof(int), hipMemcpyDeviceToHost);

    // for (int j = 0; j < resY; j++) {
    //     for (int i = 0; i < resX; i++) {
    //         printf("%d", host_img[j * resX + i]);
    //     }
    //     printf("\n");
    // }

    memcpy(img, host_img,resX * resY * sizeof(int));
    hipFree(d_img);
}
